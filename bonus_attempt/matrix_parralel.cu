
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define IM_DIM 512
#define KERNEL_DIM 7
#define THREADS_PER_BLOCK 16

typedef struct {
    double arr[KERNEL_DIM][KERNEL_DIM];
} Kernel;

typedef struct {
    unsigned int arr[IM_DIM][IM_DIM];
} Gray_Image;

// returns a kernel of a given size
Kernel blur_kernel() {
    Kernel kernel;
    for (int i=0; i < KERNEL_DIM; i++) {
        for (int j=0; j < KERNEL_DIM; j++) {
            kernel.arr[i][j] = 1.0/(KERNEL_DIM*KERNEL_DIM);
        }
    }
    return kernel;
}



    // applies a kernel to blur images 
__global__ void apply_kernel(unsigned char* input, unsigned char* output, double *kernel, int dim) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < dim && row < dim) {
        double total = 0;
        for (int i = row - KERNEL_DIM/2; i < row+KERNEL_DIM/2+1;i++){
            for (int j = col-KERNEL_DIM/2; j < col+KERNEL_DIM/2+1;j++){
                double kernel_gradient = kernel[(i - (row - KERNEL_DIM/2)) * KERNEL_DIM 
                + j - (col - KERNEL_DIM/2)];
                if(0<=i&& i<IM_DIM && 0<=j&& j<IM_DIM){
                    total += ((double)input[i*IM_DIM+j])*kernel_gradient;
                }
                else{
                    total += ((double)input[row*IM_DIM+col])*kernel_gradient;
                }
            }
        }
        output[row * dim + col] = (unsigned char)((0 <= total && total <= 255)?total:(total > 255)?255:0);
    }
}

Gray_Image make_image(){
    Gray_Image image = {.arr = {{0}}};
    for (int i = 0; i < IM_DIM; i++){
        for (int j = 0; j < IM_DIM; j++){
            image.arr[i][j] = rand()%256;
        }
    }
    return image;
}

// Prints the upper left corner of  the image, for the feel of it.
void print_image(Gray_Image image){
    for (int i = 0; i < 10; i++){
        for (int j = 0; j < 10; j++){
            printf("%d,",image.arr[i][j]);
        }
        printf(" ...\n");
    }
    printf(" ...\n");
}
// applies the kernel to a certain location
double kernel_to_pixel(Gray_Image image, int row_id, int col_id, Kernel kernel){
    double total = 0;
    for (int i = row_id - KERNEL_DIM/2; i < row_id+KERNEL_DIM/2+1;i++){
        for (int j = col_id-KERNEL_DIM/2; j < col_id+KERNEL_DIM/2+1;j++){
            double kernel_gradient = kernel.arr[i-(row_id - KERNEL_DIM/2)][j-(col_id - KERNEL_DIM/2)];
            if(0<=i&& i<IM_DIM && 0<=j&& j<IM_DIM){
                total += ((double)(image.arr[i][j]))*kernel_gradient;
            }
            else{
                total += ((double)(image.arr[row_id][col_id]))*kernel_gradient;
            }
        }
    }
    if (0 <= total && total <= 255){
        return total;
    }
    if (total > 255){
        return 255;
    }
    return 0;
}
Gray_Image apply_kernel_cpu(Gray_Image image, Kernel kernel){
    Gray_Image blured = {.arr = {{0}}};
    for (int i = 0; i < IM_DIM; i++){
        for (int j = 0; j < IM_DIM; j++){
            blured.arr[i][j] = (int)(kernel_to_pixel(image,i,j,kernel));
        }
    }
    return blured;
}
int main() {
    Gray_Image original = make_image();
    Kernel kernel = blur_kernel();
    Gray_Image blured;

    printf("Original image:\n");
    print_image(original);

    time_t g_begin = time(NULL);
    unsigned char *o_image, *b_image;
    double  *d_kernel;
    size_t image_size = sizeof(Gray_Image);
    size_t kernel_size = sizeof(Kernel);
    hipMalloc(&o_image, image_size);
    hipMalloc(&b_image, image_size);
    hipMalloc(&d_kernel, kernel_size);
    hipMemcpy(o_image, original.arr, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.arr, kernel_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 numBlocks(IM_DIM / threadsPerBlock.x, IM_DIM / threadsPerBlock.y);
    apply_kernel<<<numBlocks, threadsPerBlock>>>(o_image, b_image, d_kernel, IM_DIM);

    hipMemcpy(blured.arr, b_image, image_size, hipMemcpyDeviceToHost);

    hipFree(o_image);
    hipFree(b_image);
    hipFree(d_kernel);

    time_t gpu_time = time(NULL) - g_begin;
    printf("Blured image GPU, time: %ld s:\n",gpu_time);
    print_image(blured);

    time_t c_begin = time(NULL);
    Gray_Image blured_cpu = apply_kernel_cpu(original, kernel);
    time_t cpu_time = time(NULL) - c_begin;
    printf("Blured image CPU, time: %ld s:\n",cpu_time);
    print_image(blured_cpu);
  
    return 0;
}